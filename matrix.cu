#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 1024

__global__ void vector_matrix(int *x, int *y, int *z, int n) {
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int scan_idx;
	unsigned int ans = 0;
	for (scan_idx=0; scan_idx<n; scan_idx++) {
		ans += x[col_idx * n + scan_idx] * y[scan_idx * n + row_idx];
		__syncthreads();
	}
	z[col_idx * n + row_idx] = ans; 
}

int main(int argc, char *argv[])
{
	FILE *fpa,*fpb,*fpc,*fpx,*fpy,*fpz;
	unsigned int col_idx, row_idx;
	int* matrixA;
	int* matrixB;
	int* matrixC;
	int* matrixX;
	int* matrixY;
	int* matrixZ;
	int* matrixX_d;
	int* matrixY_d;
	int* matrixZ_d;
	int n;
        if(argc < 2) {
                n = N;
        } else {
                n = atoi(argv[1]);
        }

	matrixA = (int*)malloc(sizeof(int)*n*n);
	matrixB = (int*)malloc(sizeof(int)*n*n);
	matrixC = (int*)malloc(sizeof(int)*n*n);

        if(argc < 2) {
		for (col_idx=0; col_idx<n; col_idx++) {
			for (row_idx=0; row_idx<n; row_idx++) {
				/* matrixA[col_idx * n + row_idx] = rand() % (1024*1024); */
				/* matrixB[col_idx * n + row_idx] = rand() % (1024*1024); */
				matrixA[col_idx * n + row_idx] = 1;
				matrixB[col_idx * n + row_idx] = 1;
				matrixC[col_idx * n + row_idx] = 0;
			}
		}
        	fpa = fopen("./matrix_a.bin", "wr");
        	fpb = fopen("./matrix_b.bin", "wr");
        	fpc = fopen("./matrix_c.bin", "wr");
        	fwrite(matrixA, sizeof(int), n*n, fpa);
        	fwrite(matrixB, sizeof(int), n*n, fpb);
        	fwrite(matrixC, sizeof(int), n*n, fpc);
        	fclose(fpa);
        	fclose(fpb);
        	fclose(fpc);
	}

	matrixX = (int*)malloc(sizeof(int)*n*n);
	matrixY = (int*)malloc(sizeof(int)*n*n);
	matrixZ = (int*)malloc(sizeof(int)*n*n);
	hipMalloc(&matrixX_d, sizeof(int)*n*n);
	hipMalloc(&matrixY_d, sizeof(int)*n*n);
	hipMalloc(&matrixZ_d, sizeof(int)*n*n);

        fpx = fopen("./matrix_a.bin", "r");
        fpy = fopen("./matrix_b.bin", "r");
        fpz = fopen("./matrix_c.bin", "wr");
        fread(matrixX, sizeof(int), n*n, fpx);
        fread(matrixY, sizeof(int), n*n, fpy);

	hipMemcpy(matrixX_d, matrixX, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(matrixY_d, matrixY, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(matrixZ_d, matrixZ, sizeof(int)*n*n, hipMemcpyHostToDevice);

        int blocksize = 16;
        int gridsize = n/blocksize;
        dim3 dimGrid(gridsize,gridsize);
        dim3 dimBlock(blocksize,blocksize);
        vector_matrix<<<dimGrid,dimBlock>>>(matrixX_d,matrixY_d,matrixZ_d,n);

	hipMemcpy(matrixZ, matrixZ_d, sizeof(int)*n*n, hipMemcpyDeviceToHost);
        fwrite(matrixZ, sizeof(int), n*n, fpz);
        fclose(fpx);
        fclose(fpy);
        fclose(fpz);

	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixX);
	free(matrixY);
	free(matrixZ);
	hipFree(matrixX_d);
	hipFree(matrixY_d);
	hipFree(matrixZ_d);
}
