#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <fcntl.h>
#include <unistd.h>

#include "cufile.h"

#define N 1024

__global__ void hello(int *x) {
		unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
		printf("x:%d\n", x[col_idx]); 
}

__global__ void vector_matrix(int *x, int *y, int *z, int n) {
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int scan_idx;
	unsigned int ans = 0;
	for (scan_idx=0; scan_idx<n; scan_idx++) {
		ans += x[col_idx * n + scan_idx] * y[scan_idx * n + row_idx];
		__syncthreads();
	}
	z[col_idx * n + row_idx] = ans; 
	/* printf("z:%d\n", z[0]); */
}

int main(int argc, char *argv[])
{
	FILE *fpa,*fpb,*fpc;
	int fpx,fpy,fpz,fpzz;
	unsigned int col_idx, row_idx;
	int* matrixA;
	int* matrixB;
	int* matrixC;
	int* matrixX;
	int* matrixY;
	int* matrixZ;
	int* matrixX_d;
	int* matrixY_d;
	int* matrixZ_d;
        CUfileDescr_t cf_desc_x;
        CUfileDescr_t cf_desc_y;
        CUfileDescr_t cf_desc_z;
        CUfileHandle_t cf_handle_x;
        CUfileHandle_t cf_handle_y;
        CUfileHandle_t cf_handle_z;
	int n;
        if(argc < 2) {
                n = N;
        } else {
                n = atoi(argv[1]);
        }

	matrixA = (int*)malloc(sizeof(int)*n*n);
	matrixB = (int*)malloc(sizeof(int)*n*n);
	matrixC = (int*)malloc(sizeof(int)*n*n);

        if(argc < 2) {
		for (col_idx=0; col_idx<n; col_idx++) {
			for (row_idx=0; row_idx<n; row_idx++) {
				/* matrixA[col_idx * n + row_idx] = rand() % (1024*1024); */
				/* matrixB[col_idx * n + row_idx] = rand() % (1024*1024); */
				matrixA[col_idx * n + row_idx] = 1;
				matrixB[col_idx * n + row_idx] = 1;
				matrixC[col_idx * n + row_idx] = 0;
			}
		}
        	fpa = fopen("./matrix_a.bin", "w");
        	fpb = fopen("./matrix_b.bin", "w");
        	fpc = fopen("./matrix_c.bin", "w");
        	fwrite(matrixA, sizeof(int), n*n, fpa);
        	fwrite(matrixB, sizeof(int), n*n, fpb);
        	fwrite(matrixC, sizeof(int), n*n, fpc);
        	fclose(fpa);
        	fclose(fpb);
        	fclose(fpc);
	}

	matrixX = (int*)malloc(sizeof(int)*n*n);
	matrixY = (int*)malloc(sizeof(int)*n*n);
	matrixZ = (int*)malloc(sizeof(int)*n*n);
	hipMalloc(&matrixX_d, sizeof(int)*n*n);
	hipMalloc(&matrixY_d, sizeof(int)*n*n);
	hipMalloc(&matrixZ_d, sizeof(int)*n*n);

	cuFileDriverOpen();
        fpx = open("./matrix_a.bin", O_RDONLY | O_DIRECT);
        fpy = open("./matrix_b.bin", O_RDONLY | O_DIRECT);
        fpz = open("./matrix_c.bin", O_RDWR | O_DIRECT);
        fpzz = open("./matrix_cc.bin", O_RDWR | O_CREAT, 0777);
        cf_desc_x.handle.fd = fpx;
        cf_desc_y.handle.fd = fpy;
        cf_desc_z.handle.fd = fpz;
        cf_desc_x.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cf_desc_y.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cf_desc_z.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cuFileHandleRegister(&cf_handle_x, &cf_desc_x);
        cuFileHandleRegister(&cf_handle_y, &cf_desc_y);
        cuFileHandleRegister(&cf_handle_z, &cf_desc_z);
        cuFileBufRegister((int*)matrixX_d, sizeof(int)*n*n, 0);
        cuFileBufRegister((int*)matrixY_d, sizeof(int)*n*n, 0);
        cuFileBufRegister((int*)matrixZ_d, sizeof(int)*n*n, 0);

        cuFileRead(cf_handle_x, (int*)matrixX_d, sizeof(int)*n*n, 0, 0);
        cuFileRead(cf_handle_y, (int*)matrixY_d, sizeof(int)*n*n, 0, 0);
        cuFileRead(cf_handle_z, (int*)matrixZ_d, sizeof(int)*n*n, 0, 0);

        int blocksize = 16;
        int gridsize = n/blocksize;
        dim3 dimGrid(gridsize,gridsize);
        dim3 dimBlock(blocksize,blocksize);
        vector_matrix<<<dimGrid,dimBlock>>>(matrixX_d,matrixY_d,matrixZ_d,n);
	/* hello<<<2,4>>>(matrixX_d); */

	for(int i=0;i<n*n/512;i++) {
		cuFileWrite(cf_handle_z, matrixZ_d, sizeof(int)*512, sizeof(int)*512*i, 0);
	}
	/*
        cudaMemcpy(matrixZ, matrixZ_d, sizeof(int)*n*n, cudaMemcpyDeviceToHost); 
	printf("z:%d\n", matrixZ[0]);
	int ret=0;
	ret=pwrite(fpzz, matrixZ, sizeof(int)*n*n, 0); 
	printf("ret:%d\n",ret);
	*/

        cuFileBufDeregister((int*)matrixX_d);
        cuFileBufDeregister((int*)matrixY_d);
        cuFileBufDeregister((int*)matrixZ_d);

        close(fpx);
        close(fpy);
        close(fpz);

	cuFileDriverClose();

	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixX);
	free(matrixY);
	free(matrixZ);
	hipFree(matrixX_d);
	hipFree(matrixY_d);
	hipFree(matrixZ_d);
}
